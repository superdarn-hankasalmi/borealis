/*
Copyright 2017 SuperDARN Canada

See LICENSE for details

This file contains the CUDA code used to process the large amount of data
involved with radar receive side processing.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h> //cuComplex type and all cuCmulf/cuCaddf functions.
#include <iostream>
#include <stdint.h>

//This keeps the contained functions local to this file.
namespace {
  /**
   * @brief      Creates a new set of grid dimensions for a decimate CUDA kernel.
   *
   * @param[in]  num_samples   Number of input samples.
   * @param[in]  dm_rate       Decimation rate.
   * @param[in]  num_antennas  Number of antennas for which there are samples.
   *
   * @return     New grid dimensions for the kernel.
   */
  dim3 create_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_antennas) // REVIEW #26 no more antennas
  {
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_antennas;
    auto num_blocks_z = 1;
    std::cout << "    Grid size: " << num_blocks_x << " x " << num_blocks_y << " x "
      << num_blocks_z << std::endl;
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
  }

  /**
   * @brief      Creates a new set of block dimensions for a decimate CUDA kernel.
   *
   * @param[in]  num_taps_per_filter  Number of taps per filter.
   * @param[in]  num_freqs            Number of receive frequencies.
   *
   * @return     New block dimensions for the kernel.
   */
  dim3 create_block(uint32_t num_taps_per_filter, uint32_t num_freqs)
  {
    auto num_threads_x = num_taps_per_filter;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    std::cout << "    Block size: " << num_threads_x << " x " << num_threads_y << " x "
      << num_threads_z << std::endl;
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
  }
}

/**
 * @brief      Overloads __shfl_down to handle cuComplex.
 *
 * @param[in]  var      cuComplex value to shuffle.
 * @param[in]  srcLane  Relative lane from within the warp that should shuffle its variable down.
 * @param[in]  width    Section of the warp to shuffle. Defaults to full warp size.
 *
 * @return     Shuffled cuComplex variable.
 *
 * __shfl can only shuffle 4 bytes at time. This overload utilizes a trick similar to the below
 * link in order to shuffle 8 byte values.
 * https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
 */
__device__ inline hipComplex __shfl_down(hipComplex var, unsigned int srcLane, int width=32){
  float2 a = *reinterpret_cast<float2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<hipComplex*>(&a);
}

/**
 * @brief      Performs a parallel reduction to sum a series of values together.
 *
 * @param      data        A pointer to a set of cuComplex data to reduce.
 * @param[in]  tap_offset  The offset into the data from which to pull values.
 *
 * @return     Final sum after reduction.
 *
 * NVIDIA supplies many versions of optimized parallel reduction. This is a slightly modified
 * version of reduction #5 from NVIDIA examples.
 * /usr/local/cuda/samples/6_Advanced/reduction
 */
__device__ hipComplex parallel_reduce(hipComplex* data, uint32_t tap_offset) {
  auto filter_tap_num = threadIdx.x;
  auto num_filter_taps = blockDim.x;
  hipComplex total_sum = data[tap_offset];

  if ((num_filter_taps >= 1024) && (filter_tap_num < 512))
  {
    total_sum = hipCaddf(total_sum,data[tap_offset  + 512]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ((num_filter_taps >= 512) && (filter_tap_num < 256))
  {
    total_sum = hipCaddf(total_sum,data[tap_offset  + 256]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ((num_filter_taps >= 256) && (filter_tap_num < 128))
  {
    total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
    data[tap_offset] = total_sum;
  }

   __syncthreads();

  if ((num_filter_taps >= 128) && (filter_tap_num <  64))
  {
    total_sum = hipCaddf(total_sum, data[tap_offset  +  64]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ( filter_tap_num < 32 )
  {
    // Fetch final intermediate sum from 2nd warp
    if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
    // Reduce final warp using shuffle
    // http://docs.nvidia.com/cuda/cuda-c-programming-guide/#built-in-variables
    // __shfl_down is used an optimization in the final warp to simulatenously move
    // values from upper threads to lower threads without needing __syncthreads().
    for (int offset = warpSize/2; offset > 0; offset /= 2)
    {
      total_sum = hipCaddf(total_sum,__shfl_down(total_sum, offset));
    }
  }

  return total_sum;
}

/**
 * @brief      Performs decimation on a set of samples if the total number of filter taps for all
 *             filters is less than 1024.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each
 *                                  frequency after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able
 * process data in realtime. This algorithm will use 1 GPU thread per filter tap if there are less
 * than 1024 taps for all filters combined. Only works with power of two length filters, or a
 * filter that is zero padded to a power of two in length.
 *
 *   gridDim.x - Total number of output samples there will be after decimation.
 *   gridDim.y - Total number of antennas.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *
 *   blockDim.x - Number of filter taps in each filter.
 *   blockDim.y - Total number of filters. Corresponds to total receive frequencies.
 *
 *   threadIdx.x - Filter tap index.
 *   threadIdx.y - Filter index.
 */
__global__ void decimate1024(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna) {

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = threadIdx.y * blockDim.x + threadIdx.x;

  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples using these will be discarded as to not introduce edge effects
  hipComplex sample;
  if ((dec_sample_offset + threadIdx.x) >= samples_per_antenna) {
    sample = make_hipComplex(0.0f,0.0f);
  }
  else {
    auto final_offset = antenna_offset + dec_sample_offset + threadIdx.x;
    sample = original_samples[final_offset];
  }


  filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]);
  // Synchronizes all threads in a block, meaning 1 output sample per rx freq
  // is ready to be calculated with the parallel reduce
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {
    antenna_offset = antenna_num * samples_per_antenna/dm_rate; // REVIEW #13 gridDimx is already samples_per_antenna/dm_rate, use it instead
                                  // REPLY it is, yet I've been trying to avoid direct use of CUDA builtin types since it not obvious what they mean
    auto total_antennas = gridDim.y;
    auto freq_offset = threadIdx.y * gridDim.x * total_antennas;
    auto total_offset = freq_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;
  }
}

/**
 * @brief      Performs decimation on a set of samples if the total number of filter taps for all
 *             filters is less than 2048.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able process
 * data in realtime. This algorithm will use 1 GPU thread to process two filter taps if there are
 * less than 2048 taps for all filters combined. Intended to be used if there are more than 1024
 * total threads, as that is the max block size possible for CUDA. Only works with power of two
 * length filters, or a filter that is zero padded to a power of two in length.
 *
 *   gridDim.x - Total number of output samples there will be after decimation.
 *   gridDim.y - Total number of antennas.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *
 *   blockDim.x - Number of filter taps in each filter.
 *   blockDim.y - Total number of filters. Corresponds to total receive frequencies.
 *
 *   threadIdx.x - Filter tap index.
 *   threadIdx.y - Filter index.
 */
__global__ void decimate2048(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna)
{

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = threadIdx.y * blockDim.x + 2 * threadIdx.x;

  hipComplex sample_1;
  hipComplex sample_2;
  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples using these will be discarded as to not introduce edge effects
  if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_antenna) {
    // the case both samples are out of bounds
    sample_1 = make_hipComplex(0.0,0.0);
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_antenna - 1) {
    // the case only one sample would be out of bounds
    auto final_offset = antenna_offset + dec_sample_offset + 2*threadIdx.x;
    sample_1 = original_samples[final_offset];
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else {
    auto final_offset = antenna_offset + dec_sample_offset + 2*threadIdx.x;
    sample_1 = original_samples[final_offset];
    sample_2 = original_samples[final_offset+1];
  }


  filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]);
  filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]);

  // An additional add must happen first in this case since the parallel reduce will only
  // run on even data indices.
  filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],filter_products[tap_offset+1]);
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {
    antenna_offset = antenna_num * samples_per_antenna/dm_rate; // REVIEW #13 gridDimx is already samples_per_antenna/dm_rate, use it instead
                                  // REPLY it is, yet I've been trying to avoid direct use of CUDA builtin types since it not obvious what they mean
    auto total_antennas = gridDim.y;
    auto freq_offset = threadIdx.y * gridDim.x * total_antennas;
    auto total_offset = freq_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;
  }
}

/**
 * @brief      This function wraps the decimate1024 kernel so that it can be called from another
 *             file.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequencies.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void decimate1024_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

  auto dimGrid = create_grid(samples_per_antenna, dm_rate, num_antennas);
  auto dimBlock = create_block(num_taps_per_filter,num_freqs);
  decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
        filter_taps, dm_rate, samples_per_antenna);

}

/**
 * @brief      This function wraps the decimate2048 kernel so that it can be called from another
 *             file.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequencies.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void decimate2048_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

  auto dimGrid = create_grid(samples_per_antenna, dm_rate, num_antennas);
  auto dimBlock = create_block(num_taps_per_filter/2, num_freqs);
  decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
    filter_taps, dm_rate, samples_per_antenna);
}